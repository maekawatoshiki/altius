#include "hip/hip_runtime.h"
#include <assert.h>
#include <blis/blis.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <cudnn_backend.h>
#include <hiprand.h>
#include <stdio.h>
#include <sys/time.h>

#define ATTEMPT 10

double now_in_sec() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (double)tv.tv_sec + (double)tv.tv_usec / 1000.f / 1000.f;
}

__global__ void relu(float *x, int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = x[i] > 0.f ? x[i] : 0.f;
  }
}

__global__ void sigmoid(float *x, int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = 1.f / (1.f + expf(-x[i]));
  }
}

extern "C" void entry() {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float *lhs, *rhs, *result;
  const int m = 1000, k = 200, n = 100;
  const float alpha = 1.f, beta = 0.f;

  hipMalloc(&lhs, m * k * sizeof(float));
  hipMalloc(&rhs, k * n * sizeof(float));
  hipMalloc(&result, m * n * sizeof(float));

  {
    // fill lhs and rhs with random numbers
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateUniform(gen, lhs, m * k);
    hiprandGenerateUniform(gen, rhs, k * n);
    hiprandDestroyGenerator(gen);
  }

  for (int attempt = 0; attempt < ATTEMPT; attempt++) {
    const double start = now_in_sec();
    for (int i = 0; i < 1000; i++) {
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, rhs, n,
                  lhs, k, &beta, result, n);
      // relu<<<(m * n + 31) / 32, 32>>>(result, m * n);
      // sigmoid<<<(m * n + 31) / 32, 32>>>(result, m * n);
    }
    const double end = now_in_sec();
    printf("GPU Time: %lf[ms]\n", (end - start) * 1000.0);
  }

  float *gpu_result = (float *)malloc(m * n * sizeof(float));
  hipMemcpy(gpu_result, result, m * n * sizeof(float), hipMemcpyDeviceToHost);

  float *cpu_lhs = (float *)malloc(m * k * sizeof(float));
  float *cpu_rhs = (float *)malloc(k * n * sizeof(float));
  float *cpu_result = (float *)malloc(m * n * sizeof(float));

  hipMemcpy(cpu_lhs, lhs, m * k * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(cpu_rhs, rhs, k * n * sizeof(float), hipMemcpyDeviceToHost);

  for (int attempt = 0; attempt < ATTEMPT; attempt++) {
    const double start = now_in_sec();
    for (int i = 0; i < 1000; i++) {
      cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, alpha,
                  cpu_lhs, k, cpu_rhs, n, beta, cpu_result, n);
    }
    const double end = now_in_sec();
    printf("CPU Time: %lf[ms]\n", (end - start) * 1000.0);
  }

  for (int i = 0; i < m * n; i++) {
    const float diff = fabs(gpu_result[i] - cpu_result[i]);
    assert(diff < 1e-3);
  }

  hipFree(lhs);
  hipFree(rhs);
  hipFree(result);
}
