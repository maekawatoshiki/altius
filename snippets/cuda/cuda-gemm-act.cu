#include <stdio.h>
#include <sys/time.h>

#include <hipblas.h>
#include <hipblas.h>
#include <hipDNN.h>

double now_in_sec() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (double)tv.tv_sec + (double)tv.tv_usec / 1000.f / 1000.f;
}

extern "C" void entry() {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  const double start = now_in_sec();

  float *lhs, *rhs, *result;
  const int m = 1000, k = 200, n = 100;
  const float alpha = 1.f, beta = 0.f;

  hipMalloc(&lhs, m * k * sizeof(float));
  hipMalloc(&rhs, k * n * sizeof(float));
  hipMalloc(&result, m * n * sizeof(float));

  for (int i = 0; i < 1000; i++) {
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, lhs, m, rhs,
                k, &beta, result, m);
  }

  const double end = now_in_sec();

  printf("Time: %lf[ms]\n", (end - start) * 1000.0);

  float *host = (float *)malloc(m * n * sizeof(float));
  hipMemcpy(host, result, m * n * sizeof(float), hipMemcpyDeviceToHost);
  // for (int i = 0; i < m * n; i++) {
  //   printf("%f ", host[i]);
  // }
}
